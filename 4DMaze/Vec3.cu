#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"

#include <glm/glm.hpp>
#include <vector>
#include <string>
#include <sstream>

class Vec3 {
public:
	double u;
	double v;
	double t;
	__host__ __device__ Vec3(double u, double v, double t) : u(u), v(v), t(t) {}
	Vec3(glm::vec3 v) : u(v.x), v(v.y), t(v.z) {}
	Vec3(std::vector<double> components) : u(components[0]), v(components[1]),
		t(components[2]) {}

	__device__ Vec3 operator+(const Vec3 o) const {
		return Vec3(u + o.u, v + o.v, t + o.t);
	}

	Vec3 operator-(const Vec3 o) const {
		return Vec3(u - o.u, v - o.v, t - o.t);
	}

	__device__ friend Vec3 operator*(double lhs, const Vec3 rhs) {
		return Vec3(lhs * rhs.u, lhs * rhs.v, lhs * rhs.t);
	}

	Vec3 operator/(double o) {
		return Vec3(u / o, v / o, t / o);
	}

	operator std::string() {
		std::stringstream res;
		res << u << ' ' << v << ' ' << t;
		return res.str();
	}

	double dot(const Vec3 o) const {
		return u * o.u + v * o.v + t * o.t;
	}

	Vec3 cross(const Vec3 o) const {
		return Vec3(v * o.t - t * o.v, t * o.u - u * o.t, u * o.v - v * o.u);
	}

	double magnitude() const {
		return sqrt(u * u + v * v + t * t);
	}

	Vec3 normalize() const {
		float mag = magnitude();
		return Vec3(u / mag, v / mag, t / mag);
	}
};